#include <nervCUDA.h>

#include <nerv_kernels.h>

extern "C" {

void costFunc(unsigned int nl, unsigned int* lsizes, unsigned int nsamples, 
	double* nn_params, double* X, double* yy, double lambda, double* inputs, double& J)
{
	// Allocate the device memory:
	size_t size;
	hipError_t err;

	size = nl * sizeof(unsigned int);
	double* d_lsizes = NULL;
	err = hipMalloc(&d_lsizes, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc lsizes: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_lsizes, lsizes, size, hipMemcpyHostToDevice);

	// Compute the total number of parameters in this network:
	unsigned int np = 0;
	unsigned int nt = nl-1; // number of matrices evolved.

	for(unsigned int i=0;i<nt;++i) {
		np += lsizes[i+1]*(lsizes[i]+1);
	}

	size = np * sizeof(double);
	double* d_params = NULL;
	err = hipMalloc(&d_params, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc params: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_params, nn_params, size, hipMemcpyHostToDevice);

	// Prepare the X matrix:
	size = sizeof(double) * nsamples * lsizes[0];
	double* d_X = NULL;
	err = hipMalloc(&d_X, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc X: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);


	// Prepare the input data:
	// the size of each input matrix is lsize[i+1]*nsamples;
	// and we need input 0 to nt-1, inclusive.
	// So that's nl input matrices.
	unsigned int count = 0;
	for(unsigned int i=0;i<nt;++i) {
		count += lsizes[i+1];
	}

	size = nsamples * count * sizeof(double);
	size_t input_size = size;
	double* d_inputs = NULL;
	err = hipMalloc(&d_inputs, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc inputs: "<<hipGetErrorString(err));
	}
	hipMemset(d_inputs,0,size); // This is needed for debugging only.

	// Copy the label matrix:	
	size = nsamples * lsizes[nt] * sizeof(double);
	double* d_yy = NULL;
	err = hipMalloc(&d_yy, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc yy: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_yy, yy, size, hipMemcpyHostToDevice);


	// offset used to locate the theta_i matrix in the d_params array.
	unsigned int theta_offset = 0;

	// Offset used for the z(i) matrix on iteration i
	unsigned int input_offset = 0;

	unsigned int next_input_offset = 0; //nsamples*lsizes[1];

	double reg_correction = 0.0;
	double* tptr = nn_params;
	double rval;

  for(unsigned int i=0; i<nt;++i) {
  	// We compute the activation and input values for the given layer:

  	// The kernel compute the values of zi and a(i+1) 
  	// (note that the value or a(0) is already loaded in the Activation vector).
  	// even if we compute the a(i+1) matrix we actually discard completely the first column
  	// in this matrix (colu of intercept terms). As a result we just need to mapped the GPU grid to
  	// the dimension of of the sub z(i) matrix (which is transposed.)
  	// THe dimensions for z(i) are: lsize(i+1) * nsamples
  	// When this is transposed we get: nsamples * lsize(i+1);
		unsigned int nrows = lsizes[i+1];
		unsigned int ncolT = lsizes[i]; // we remove 1 here because we consider the intercept row as "virtual" in our calculation.
		unsigned int ncols = nsamples;

		for(unsigned int j=0;j<nrows;++j) {
			rval = (*tptr++);
			reg_correction += rval*rval;
		}
		tptr += nrows*ncolT;

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((BLOCK_SIZE + ncols-1)/BLOCK_SIZE, (BLOCK_SIZE + nrows-1)/BLOCK_SIZE);

		// Also we will need access to the theta_i matrix so we need to keep track of its global offset in the
		// network parameters array.
		// logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");
		ComputeActivation<<<dimGrid, dimBlock>>>(theta_offset, input_offset, next_input_offset,
			nrows, ncols, ncolT, d_params, d_inputs, d_X);

		// update the offsets:
		theta_offset += lsizes[i+1]*(lsizes[i]+1);
		input_offset = next_input_offset;
		next_input_offset += nrows*ncols;
  }

  // Here we can compute the cost now:
  // The hx matrix is mapped to the last z matrix. eg at i=nt-1
  // So its dimensions are lsizes[nt-1+1] * nsamples = lsizes[nl-1] * nsamples
  // same dimensions for the yy matrix, and we want to perform reduction other those 2 matrices
	J = 0.0;
	count = nsamples*lsizes[nt];
	reduction_cost_device(d_inputs + input_offset, d_yy, count, J);

	J /= (double)nsamples;

	double Jreg = 0.0;
	reduction_cost_reg_device(d_params, np, Jreg);
	Jreg -= reg_correction;
	
	J += (Jreg*lambda)/(2.0*nsamples);

	// Read inputs from device memory
	err = hipMemcpy(inputs, d_inputs, input_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess) {
		logDEBUG("CUDA reading inputs: "<<hipGetErrorString(err));
	}

	// Free device memory
	hipFree(d_lsizes);
	hipFree(d_params);
	hipFree(d_inputs);	
	hipFree(d_yy);	
	hipFree(d_X);	
}

}