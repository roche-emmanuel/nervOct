#include "hip/hip_runtime.h"
#include <nervCUDA.h>

#include <hip/hip_runtime.h>
#include <nerv_kernels.h>

__global__ void ComputeGradient(unsigned int theta_offset, unsigned int input_offset,  unsigned int delta_offset, unsigned int grad_offset,
	unsigned int nrows, unsigned int ncols, unsigned int niter, double* nn_params, double* inputs, double* deltas, double* grads, double lambda) 
{
  double CValue = 0;

  int row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
  int col = blockIdx.x*BLOCK_SIZE + threadIdx.x;

  __shared__ double As[BLOCK_SIZE][BLOCK_SIZE+1]; // Adding +1 to avoid shared memory bank conflict
  __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE+1];

  int xx, yy;
  for (int k = 0; k < (BLOCK_SIZE + niter - 1)/BLOCK_SIZE; k++) {

  	// Here we try to access the A matrix data in a coaleased way:
  	// keeping in mind that A is row major. So we need to read A per column
  	// while the threads in the wrap are (probably) organized by row.
  	// So we invert the roles palyed by threadIdx.x and threadIdx.y.
  	xx = k*BLOCK_SIZE + threadIdx.y;
  	yy = blockIdx.y*BLOCK_SIZE + threadIdx.x;
		if (xx < niter && yy < nrows) 
		 	As[threadIdx.x][threadIdx.y] = deltas[delta_offset + xx*nrows + yy];
		else
			As[threadIdx.x][threadIdx.y] = 0.0;


		// Same for the B matrix, we need to invert the x and y coords:
		xx = blockIdx.x*BLOCK_SIZE + threadIdx.y;
		yy = k*BLOCK_SIZE + threadIdx.x;

		if (yy < niter && xx < ncols) {
			// B(r,c)==0 if c==0 or B(r,c)=z_T(r,c-1)= z(c-1,r)
			Bs[threadIdx.x][threadIdx.y] = (xx==0 ? 1.0 : inputs[input_offset + (ncols-1)*yy + xx-1]); //inputs[input_offset + (ncols-1)*yy + xx-1 ]; // memory access is coalesced, nothing to change.
		}
		else
			Bs[threadIdx.x][threadIdx.y] = 0.0;

		__syncthreads();

		for (int n = 0; n < BLOCK_SIZE; ++n) 
			CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

		__syncthreads();
  }

  if (row < nrows && col < ncols) {
  	int index = nrows*col+row;
    double reg = (col==0 ? 0.0 : nn_params[theta_offset + index]);
    CValue += lambda*reg;

  	grads[grad_offset + index] = CValue/niter;
  }

	// // This operation is basically a matrix multiplication with transposition on A:
 //  double gval = 0.0;

 //  int row = blockIdx.y*BLOCK_SIZE + threadIdx.y;
 //  int col = blockIdx.x*BLOCK_SIZE + threadIdx.x;

 //  __shared__ double As[BLOCK_SIZE][BLOCK_SIZE+1];
 //  __shared__ double Bs[BLOCK_SIZE][BLOCK_SIZE+1];

 //  // So we want to compute the value d(row,col);
 //  int xx, yy;
 //  for (int k = 0; k < (BLOCK_SIZE + niter - 1)/BLOCK_SIZE; k++) {

 //  	// THe first matrix is delta[i] and it is not transposed:
 //  	xx = k*BLOCK_SIZE + threadIdx.x;
 //  	yy = blockIdx.y*BLOCK_SIZE + threadIdx.y;

 //  	// Need to access A(yy,xx)
	// 	if (xx < niter && yy < nrows) 
	// 	 	As[threadIdx.x][threadIdx.y] = 1.0; //deltas[delta_offset + xx*nrows + yy];
	// 	else
	// 		As[threadIdx.x][threadIdx.y] = 0.0;

	// 	// The second matrix contains a transposition:
	// 	xx = blockIdx.x*BLOCK_SIZE + threadIdx.x; // = col
	// 	yy = k*BLOCK_SIZE + threadIdx.y;

	// 	// We want to retrieve the value of B(yy,xx)
	// 	if (yy < niter && xx < ncols)
	// 		// B(r,c)==0 if c==0 or B(r,c)=z_T(r,c-1)= z(c-1,r)
	// 		Bs[threadIdx.x][threadIdx.y] = 1.0; //xx==0 ? 1.0 : inputs[input_offset + (ncols-1)*yy + xx-1 ]; // memory access is coalesced, nothing to change.
	// 	else
	// 		Bs[threadIdx.x][threadIdx.y] = 0.0;

	// 	__syncthreads();

	// 	for (int n = 0; n < BLOCK_SIZE; ++n) 
	// 		gval += As[threadIdx.y][n] * Bs[n][threadIdx.x];

	// 	__syncthreads();
 //  }

 //  if (row < nrows && col < ncols) {
 //  	// We should also compute the regularization term:
 //  	int index = nrows*col+row;
 //    double reg = (col==0 ? 0.0 : nn_params[theta_offset + index]);
 //    // gval += lambda*reg;

 //  	grads[grad_offset + index] = niter; //gval; //gval; //gval/(double)niter; //(niter==nsamples)
 // }
}
