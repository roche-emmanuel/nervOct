#include <nervCUDA.h>

#include <hip/hip_runtime.h>
#include <nerv_kernels.h>
#include <iostream>

#define logDEBUG(msg) std::cout << msg << std::endl;

extern "C" {

void multiplyMatrices(unsigned int nrowA, unsigned int ncolA, const double* A,
    unsigned int nrowB, unsigned int ncolB, const double* B, double* C, bool tpA, bool tpB)
{
	// Allocate the device memory:
	size_t size;
	hipError_t err;

	size = nrowA * ncolA * sizeof(double);
	double* d_A = NULL;
	err = hipMalloc(&d_A, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc A: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	size = nrowB * ncolB * sizeof(double);
	double* d_B = NULL;
	err = hipMalloc(&d_B, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc B: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	size = (tpA ? ncolA : nrowA) * (tpB ? nrowB : ncolB) * sizeof(double);
	double* d_C = NULL;
	err = hipMalloc(&d_C, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc C: "<<hipGetErrorString(err));
	}
	// hipMemcpy(d_B, B, size, hipMemcpyHostToDevice); // no need to set this.

	// Call the kernel directly:
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((BLOCK_SIZE + (tpB ? nrowB : ncolB)-1)/BLOCK_SIZE, (BLOCK_SIZE + (tpA ? ncolA : nrowA)-1)/BLOCK_SIZE);
	// logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");

	if(tpA) {
		MatMulKernelTpA<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}
	else if(tpB) {
		MatMulKernelTpB<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}
	else {
		MatMulKernel<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}

	// Read C from device memory
	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// logDEBUG("Copy C off of device: "<<hipGetErrorString(err));

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

void costFunc(unsigned int nl, unsigned int* lsizes, unsigned int nsamples, 
	double* nn_params, double* X, double* yy, double lambda,
	double* activation, double* inputs)
{
	// Allocate the device memory:
	size_t size;
	hipError_t err;

	size = nl * sizeof(unsigned int);
	double* d_lsizes = NULL;
	err = hipMalloc(&d_lsizes, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc lsizes: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_lsizes, lsizes, size, hipMemcpyHostToDevice);

	// Compute the total number of parameters in this network:
	unsigned int np = 0;
	unsigned int nt = nl-1; // number of matrices evolved.

	for(unsigned int i=0;i<nt;++i) {
		np += lsizes[i+1]*(lsizes[i]+1);
	}

	size = np * sizeof(double);
	double* d_params = NULL;
	err = hipMalloc(&d_params, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc params: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_params, nn_params, size, hipMemcpyHostToDevice);

	// Prepare the activation data:
	// the size of each activation matrix is nsamples*(lsize[i]+1);
	// and we need activation 0 to nt-1, inclusive.
	// So that's nl activation matrices.
	unsigned int count = 0;
	for(unsigned int i=0;i<nl;++i) {
		count += lsizes[i]+1;
	}

	size = nsamples * count * sizeof(double);
	size_t act_size = size;
	double* d_activation = NULL;
	err = hipMalloc(&d_activation, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc activation: "<<hipGetErrorString(err));
	}
	hipMemset(d_activation,0,size); // This is needed for debugging only.

	double* ptr = d_activation;
	
	// Coyp the intercept vector:
	// hipMemcpy(ptr, intercept, nsamples, hipMemcpyHostToDevice); // No need to copy that.
	ptr += nsamples;

	// size of the X matrix:
	size = sizeof(double) * nsamples * lsizes[0];
	hipMemcpy(ptr, X, size, hipMemcpyHostToDevice);

	// Prepare the input data:
	// the size of each input matrix is lsize[i+1]*nsamples;
	// and we need input 0 to nt-1, inclusive.
	// So that's nl input matrices.
	count = 0;
	for(unsigned int i=0;i<nt;++i) {
		count += lsizes[i+1];
	}

	size = nsamples * count * sizeof(double);
	size_t input_size = size;
	double* d_inputs = NULL;
	err = hipMalloc(&d_inputs, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc inputs: "<<hipGetErrorString(err));
	}
	hipMemset(d_inputs,0,size); // This is needed for debugging only.

	// Copy the label matrix:	
	size = nsamples * lsizes[nt] * sizeof(double);
	double* d_yy = NULL;
	err = hipMalloc(&d_yy, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc yy: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_yy, yy, size, hipMemcpyHostToDevice);


	// offset used to locate the theta_i matrix in the d_params array.
	unsigned int theta_offset = 0;
	
	// Offset used for the activation matrix i on iteration i.
	unsigned int act_offset = 0;

	// Offset used for the activation matrix (i+1) on iteration i:
	// Note that the activation matrix has a size of nsamples*(lsizes[i]+1)
	unsigned int next_act_offset = nsamples*(lsizes[0]+1);

	// Offset used for the z(i) matrix on iteration i
	unsigned int input_offset = 0;

  for(unsigned int i=0; i<nt;++i) {
  	// We compute the activation and input values for the given layer:

  	// The kernel compute the values of zi and a(i+1) 
  	// (note that the value or a(0) is already loaded in the Activation vector).
  	// even if we compute the a(i+1) matrix we actually discard completely the first column
  	// in this matrix (colu of intercept terms). As a result we just need to mapped the GPU grid to
  	// the dimension of of the sub z(i) matrix (which is transposed.)
  	// THe dimensions for z(i) are: lsize(i+1) * nsamples
  	// When this is transposed we get: nsamples * lsize(i+1);
		unsigned int nrows = lsizes[i+1];
		unsigned int ncolT = lsizes[i]+1;
		unsigned int ncols = nsamples;

		dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
		dim3 dimGrid((BLOCK_SIZE + ncols-1)/BLOCK_SIZE, (BLOCK_SIZE + nrows-1)/BLOCK_SIZE);

		// Also we will need access to the theta_i matrix so we need to keep track of its global offset in the
		// network parameters array.
		// logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");
		ComputeActivation<<<dimGrid, dimBlock>>>(theta_offset, act_offset, next_act_offset, input_offset, 
			nrows, ncols, ncolT, d_params,d_activation,d_inputs);

		// update the offsets:
		theta_offset += lsizes[i+1]*(lsizes[i]+1);
		act_offset = next_act_offset;
		next_act_offset += nsamples*(lsizes[i+1]+1);
		input_offset += nsamples*lsizes[i+1];
  }

	// Read activations from device memory
	err = hipMemcpy(activation, d_activation, act_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess) {
		logDEBUG("CUDA reading activation: "<<hipGetErrorString(err));
	}

	// Read inputs from device memory
	err = hipMemcpy(inputs, d_inputs, input_size, hipMemcpyDeviceToHost);
	if(err!=hipSuccess) {
		logDEBUG("CUDA reading inputs: "<<hipGetErrorString(err));
	}

	// update the intercept values in the activation array:
	ptr = activation;
	for(unsigned int i=0;i<nl;++i) {
		for(unsigned int j=0;j<nsamples;++j) {
			(*ptr++) = 1.0;
		}
		ptr += nsamples*(lsizes[i]);
	}

	// Free device memory
	hipFree(d_lsizes);
	hipFree(d_params);
	hipFree(d_activation);	
	hipFree(d_inputs);	
	hipFree(d_yy);	
}

}
