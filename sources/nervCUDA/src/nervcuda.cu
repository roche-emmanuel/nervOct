#include <nervCUDA.h>

#include <hip/hip_runtime.h>
#include <nerv_kernels.h>
#include <iostream>

extern "C" {

void multiplyMatrices(unsigned int nrowA, unsigned int ncolA, const double* A,
    unsigned int nrowB, unsigned int ncolB, const double* B, double* C, bool tpA, bool tpB)
{
	// Allocate the device memory:
	size_t size;
	hipError_t err;

	size = nrowA * ncolA * sizeof(double);
	double* d_A = NULL;
	err = hipMalloc(&d_A, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc A: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

	size = nrowB * ncolB * sizeof(double);
	double* d_B = NULL;
	err = hipMalloc(&d_B, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc B: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	size = (tpA ? ncolA : nrowA) * (tpB ? nrowB : ncolB) * sizeof(double);
	double* d_C = NULL;
	err = hipMalloc(&d_C, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc C: "<<hipGetErrorString(err));
	}
	// hipMemcpy(d_B, B, size, hipMemcpyHostToDevice); // no need to set this.

	// Call the kernel directly:
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((BLOCK_SIZE + (tpB ? nrowB : ncolB)-1)/BLOCK_SIZE, (BLOCK_SIZE + (tpA ? ncolA : nrowA)-1)/BLOCK_SIZE);
	// logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");

	if(tpA) {
		MatMulKernelTpA<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}
	else if(tpB) {
		MatMulKernelTpB<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}
	else {
		MatMulKernel<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}

	// Read C from device memory
	err = hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	// logDEBUG("Copy C off of device: "<<hipGetErrorString(err));

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

}
