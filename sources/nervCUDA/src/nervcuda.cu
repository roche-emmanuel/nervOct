#include <nervCUDA.h>

#include <hip/hip_runtime.h>
#include <nerv_kernels.h>
#include <iostream>

extern "C" {

void multiplyMatrices(unsigned int nrowA, unsigned int ncolA, const double* A,
    unsigned int nrowB, unsigned int ncolB, const double* B, double* C, bool tpA, bool tpB)
{
	// Allocate the device memory:
	size_t size;

	size = nrowA * ncolA * sizeof(double);
	double* d_A = NULL;
	checkCudaErrors(hipMalloc(&d_A, size));
	checkCudaErrors(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));

	size = nrowB * ncolB * sizeof(double);
	double* d_B = NULL;
	checkCudaErrors(hipMalloc(&d_B, size));
	checkCudaErrors(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

	size = (tpA ? ncolA : nrowA) * (tpB ? nrowB : ncolB) * sizeof(double);
	double* d_C = NULL;
	checkCudaErrors(hipMalloc(&d_C, size));
	// hipMemcpy(d_B, B, size, hipMemcpyHostToDevice); // no need to set this.

	// Call the kernel directly:
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((BLOCK_SIZE + (tpB ? nrowB : ncolB)-1)/BLOCK_SIZE, (BLOCK_SIZE + (tpA ? ncolA : nrowA)-1)/BLOCK_SIZE);
	// logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");

	if(tpA) {
		MatMulKernelTpA<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}
	else if(tpB) {
		MatMulKernelTpB<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}
	else {
		MatMulKernel<<<dimGrid, dimBlock>>>(nrowA, ncolA, d_A, nrowB, ncolB, d_B, d_C);
	}

	// Read C from device memory
	checkCudaErrors(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
	// logDEBUG("Copy C off of device: "<<hipGetErrorString(err));

	// Free device memory
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}

}
