#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

#include <hiprand/hiprand_kernel.h>

__device__ float random_float(hiprandState *states, int rid)
{
  hiprandState rState = states[rid];
  float res = hiprand_uniform(&rState);
  states[rid] = rState;
  return res;
}


template <typename T, unsigned int blockSize>
__global__ void ComputeActivationWithDropout
(BPComputeTraits<T> traits)
{
  unsigned int nrows = traits.nrows;
  unsigned int ncols = traits.ncols;
  unsigned int ncolT = traits.niter;
  hiprandState *states = traits.randStates;

  // Compute the index to retrieve the rand state:
  int rid = blockSize * threadIdx.y + threadIdx.x;

  // Retrieve the dropout threshold:
  T drop = traits.layer_dropout;

  // Note that we assume here that the matrix coefficient are stored in row major order:
  // eg Aelem(i,jl) = A[j*nrowA+i]
  int row = blockIdx.y * blockSize + threadIdx.x;
  int col = blockIdx.x * blockSize + threadIdx.y;

  __shared__ T As[blockSize][blockSize + 1];
  __shared__ T Bs[blockSize][blockSize + 1];

  int xx, yy;

  // we can already add the element on the first row of theta_i to this element value:
  // but note that this element should be multiplied with the desired bias:
  // Here we can use the wbias array, to decide if the bias unit is activated
  // for that sample or not:
  T zval = traits.params[traits.theta_offset + row] * traits.wbias[traits.wbias_offset + col];

  // Here we compute the product theta_i * a_i^T
  for (int k = 0; k < (blockSize + ncolT - 1) / blockSize; k++)
  {

    xx = k * blockSize + threadIdx.y;
    yy = blockIdx.y * blockSize + threadIdx.x;

    if (xx < ncolT && yy < nrows)
      // Note here that we should NOT use the first row of theta_i in those computation:
      // That row elemtn is already added to the zval value (matching the "virtual" 1 row
      // on top of the z_i matrix when used as activation.)
      As[threadIdx.x][threadIdx.y] = traits.params[traits.theta_offset + (xx + 1) * nrows + yy];
    else
      As[threadIdx.x][threadIdx.y] = 0.0;


    if (traits.next_input_offset == 0)
    {
      // In that case we need to retrieve the data from the X matrix.
      // actually we need the data from X^T.
      xx = blockIdx.x * blockSize + threadIdx.x;
      yy = k * blockSize + threadIdx.y;

      if (xx < ncols && yy < ncolT)
        Bs[threadIdx.y][threadIdx.x] = traits.X[yy * ncols + xx];
      else
        Bs[threadIdx.y][threadIdx.x] = 0.0;
    }
    else
    {
      xx = blockIdx.x * blockSize + threadIdx.y;
      yy = k * blockSize + threadIdx.x;

      if (yy < ncolT && xx < ncols)
        Bs[threadIdx.x][threadIdx.y] = traits.inputs[traits.input_offset + xx * ncolT + yy];
      else
        Bs[threadIdx.x][threadIdx.y] = 0.0;
    }

    __syncthreads();

    for (int n = 0; n < blockSize; ++n)
      zval += As[threadIdx.x][n] * Bs[n][threadIdx.y];

    __syncthreads();
  }

  if (row < nrows && col < ncols)
  {
    // check if we need to keep this unit activated or not
    // depending on the current dropout threshold:
    if (random_float(states, rid) <= drop)
    {
      // compute the sigmoid of the value:
      zval = 1.0 / (1.0 + exp(-zval * traits.wmult));
    }
    else
    {
      zval = 0.0; // desactivate this unit.
    }

    // we just computed the value z_i(row,col), now we store it:
    traits.inputs[traits.next_input_offset + nrows * col + row] = zval;
  }

}

template __global__ void ComputeActivationWithDropout<double>(BPComputeTraits<double> traits);
template __global__ void ComputeActivationWithDropout<float>(BPComputeTraits<float> traits);
