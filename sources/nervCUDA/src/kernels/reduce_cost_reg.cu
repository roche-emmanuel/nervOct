#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

/*
	Method used to evaluate the cost function when starting from the hx and yy matrices.
*/
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void ReduceCostReg(T *g_idata, T* g_regw, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
    		T val = g_idata[i];
        mySum += val*val*g_regw[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) {
        	val = g_idata[i+blockSize];
        	mySum += val*val*g_regw[i+blockSize];
        }
         
        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template<typename T>
void reduce_cost_reg_launcher(int size, int threads, int blocks,
       int whichKernel, T *d_idata, T* d_regw, T *d_odata)
{
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

  // choose which of the optimized versions of reduction to launch
  if (isPow2(size))
  {
      switch (threads)
      {
          case 512:
              ReduceCostReg<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 256:
              ReduceCostReg<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 128:
              ReduceCostReg<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 64:
              ReduceCostReg<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 32:
              ReduceCostReg<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 16:
              ReduceCostReg<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  8:
              ReduceCostReg<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  4:
              ReduceCostReg<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  2:
              ReduceCostReg<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  1:
              ReduceCostReg<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;
      }
  }
  else
  {
      switch (threads)
      {
          case 512:
              ReduceCostReg<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 256:
              ReduceCostReg<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 128:
              ReduceCostReg<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 64:
              ReduceCostReg<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 32:
              ReduceCostReg<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case 16:
              ReduceCostReg<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  8:
              ReduceCostReg<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  4:
              ReduceCostReg<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  2:
              ReduceCostReg<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;

          case  1:
              ReduceCostReg<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_regw, d_odata, size);
              break;
      }
  }
}

template<typename T>
void reduce_cost_reg_device(T* d_params, T* d_regw, unsigned int n, T& output)
{
  int maxThreads = 256;
  int maxBlocks = 64;
  int whichKernel = 6;
  // bool cpuFinalReduction = false;
  int cpuFinalThreshold = 1;

  int numBlocks = 0;
  int numThreads = 0;
  getNumBlocksAndThreads(whichKernel, n, maxBlocks, maxThreads, numBlocks, numThreads);

  // Allocate output array:
  size_t size = numBlocks*sizeof(T);
  T* d_odata = NULL;
  checkCudaErrors(hipMalloc(&d_odata, size));

  // Allocate mem for the result on host side
  T *h_odata = (T *) malloc(numBlocks*sizeof(T));

  T gpu_result = 0.0;
  bool needReadBack = true;

  // execute the kernel
  reduce_cost_reg_launcher(n, numThreads, numBlocks, whichKernel, d_params, d_regw, d_odata);

  // sum partial block sums on GPU
  int s=numBlocks;
  int kernel = whichKernel;

  while (s > cpuFinalThreshold)
  {
      int threads = 0, blocks = 0;
      getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);

      reduce_sum_launcher(s, threads, blocks, kernel, d_odata, d_odata);

      if (kernel < 3)
      {
          s = (s + threads - 1) / threads;
      }
      else
      {
          s = (s + (threads*2-1)) / (threads*2);
      }
  }

  if (s > 1)
  {
      // copy result from device to host
      checkCudaErrors(hipMemcpy(h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost));

      for (int i=0; i < s; i++)
      {
          gpu_result += h_odata[i];
      }

      needReadBack = false;
  }

  if (needReadBack)
  {
      // copy final sum from device to host
      checkCudaErrors(hipMemcpy(&gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost));
  }

  // store the result:
  output = gpu_result;

  // Free host memory:
  free(h_odata);

  // Free device memory
  checkCudaErrors(hipFree(d_odata));
}

template<typename T>
void _reduce_cost_reg(T* params, T* regweights, unsigned int n, T& output)
{
  // Allocate the params array:
  size_t size = n * sizeof(T);
  T* d_idata = NULL;
  checkCudaErrors(hipMalloc(&d_idata, size));
  checkCudaErrors(hipMemcpy(d_idata, params, size, hipMemcpyHostToDevice));

  T* d_regw = NULL;
  checkCudaErrors(hipMalloc(&d_regw, size));
  checkCudaErrors(hipMemcpy(d_regw, regweights, size, hipMemcpyHostToDevice));

  reduce_cost_reg_device(d_idata,d_regw,n,output);

  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_regw));
}


extern "C" {

void reduce_cost_reg(double* params, double* regweights, unsigned int n, double& output)
{
  _reduce_cost_reg(params, regweights, n, output);
}

void reduce_cost_reg_f(float* params, float* regweights, unsigned int n, float& output)
{
  _reduce_cost_reg(params, regweights, n, output);
}

}
