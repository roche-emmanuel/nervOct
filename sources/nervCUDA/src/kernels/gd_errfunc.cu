#include <nervCUDA.h>
#include <nerv_kernels.h>

template<typename T, unsigned int blockSize>
void gd_errfunc_device(unsigned int nl, unsigned int np, unsigned int* lsizes, unsigned int nsamples,
	T* d_params, T* d_X, T* d_yy, T lambda, T* J, T* d_grads, T* d_deltas, T* d_inputs, T* d_regw, T bias, hipStream_t stream)
{
	unsigned int nt = nl-1; // number of matrices evolved.

	BPComputeTraits<T> traits;
	traits.params = d_params;
	traits.inputs = d_inputs;
	traits.deltas = d_deltas;
	traits.grads = d_grads;
	traits.yy = d_yy;
	traits.X = d_X;
	traits.bias = bias;
	traits.lambda = lambda;

	T* wmults = nullptr;

  traits.input_offset = nn_activation_device(nl, lsizes, nsamples, d_params, d_X, d_inputs, bias, wmults, stream);

  T* d_hx = d_inputs + traits.input_offset;

  // Here we can compute the cost now:
  // but only if requested:
  if(J) {
	  // The hx matrix is mapped to the last z matrix. eg at i=nt-1
	  // So its dimensions are lsizes[nt-1+1] * nsamples = lsizes[nl-1] * nsamples
	  // same dimensions for the yy matrix, and we want to perform reduction other those 2 matrices
		*J = 0.0;
		unsigned int count = nsamples*lsizes[nt];
		reduce_cost_device(d_hx, d_yy, count, *J, stream);
		// CHECK_KERNEL()

		*J /= (T)nsamples;

		T Jreg = 0.0;
		reduce_cost_reg_device(d_params, d_regw, np, Jreg, stream);
		// CHECK_KERNEL()

		*J += (T)((Jreg*lambda)/(2.0*nsamples));
  }

  if(!d_grads) {
  	// we don't need to compute the gradients.
  	return;
  }
  
	// Prepare the computation of the delta matrices in reverse order:

	// remove the last theta matrix size from the theta offset so that we can use
	// that offset to retrieve the proper theta matrix:
	// theta_offset -= lsizes[nt]*(lsizes[nt-1]+1);
	traits.theta_offset = np - lsizes[nt]*(lsizes[nt-1]+1);

	// initially the input_offset is pointing on the hx matrix which is z(nt-1) with our convention (eg. z(0) is not in the array.)
	// But the first one we will need is actually the one before that: z(nt-2)
	// So we need to update the offset, and remove the size of the matrix z(nt-2) ! (pointer is at the beginning of z(nt-1))
	// Note: This is now done inside the loop:
	// input_offset -= lsizes[nt-1]*nsamples;

	// Prepare the offset for the gradient array:
	// keep in mind we start with the latest theta matrix:
	traits.grad_offset = np - lsizes[nt]*(lsizes[nt-1]+1);

	for(unsigned int i=nt;i>0;--i) {
		traits.nrows = lsizes[i];
		traits.ncols = nsamples;
		traits.niter = lsizes[i+1];
		unsigned int count = traits.nrows*traits.ncols;

		dim3 dimBlock(blockSize, blockSize);
		dim3 dimGrid((blockSize + traits.ncols-1)/blockSize, (blockSize + traits.nrows-1)/blockSize);

		if(i==nt) {
			// we should just copy the difference of hx and yy into the z matrix.
			// CHECK_KERNEL()
			// InitLastDelta<<<dimGrid, dimBlock, 0, stream>>>(input_offset, nrows, ncols, d_deltas, d_inputs, d_yy);
			InitLastDelta<<<dimGrid, dimBlock, 0, stream>>>(traits);
			// CHECK_KERNEL()
		}
		else {
			// We compute the delta from the previous delta:
			// We start this computation for delta(nt-1). this matrix is build from theta(nt-1) and delta(nt).
			// also in the process we use the input matrix z(nt-2)
			// ComputeDelta<<<dimGrid, dimBlock, 0, stream>>>(theta_offset, input_offset, delta_offset, next_delta_offset, nrows, ncols, niter, d_params, d_inputs, d_deltas);
			ComputeDelta<<<dimGrid, dimBlock, 0, stream>>>(traits);
			// CHECK_KERNEL()

			// once the computation is done for that layer we move to the previous layer:
			traits.theta_offset -= lsizes[i]*(lsizes[i-1]+1);
		}

		traits.delta_offset = traits.next_delta_offset;
		traits.next_delta_offset += count;

		// At this point we have the previous theta matrix (eg. theta(i-1) pointed by theta_offset. (both when i=nt and i<nt).
		// and thats the matrix we need to compute the gradient values.
		// the gradient mat has the same size as the current theta matrix.
		// similarly, the input_offset is pointing on z(i-2) which is the one we need to perform the computation too.
		// and delta_offset points to the delta matrix we just wrote (eg. delta(i)).
		traits.nrows = lsizes[i];
		traits.ncols = lsizes[i-1]+1;
		traits.niter = nsamples;
		count = traits.nrows*traits.ncols;

		// Compute the gradient:
		dimBlock = dim3(blockSize, blockSize);
		dimGrid = dim3((blockSize + traits.ncols-1)/blockSize, (blockSize + traits.nrows-1)/blockSize);

    traits.input_offset -= lsizes[i-1]*nsamples; // we remove the size of the next delta matrix to be computed. which is also the size of the next z matrix we will use.
		// logDEBUG("GPU: Gradient at i="<<i<<" of size "<< nrows <<" x " << ncols<<", offset="<<grad_offset<<", input_offset="<<input_offset<<", nsamples="<<nsamples);

		// ComputeGradient<<<dimGrid, dimBlock, 0, stream>>>(theta_offset, input_offset, delta_offset, grad_offset, nrows, ncols, niter, d_X, d_params, d_inputs, d_deltas, d_grads, lambda, bias);
		ComputeGradient<<<dimGrid, dimBlock, 0, stream>>>(traits);
		// CHECK_KERNEL()

		// update the gradient offset by removing the size of the next gradient matrix to be computed:
		// except for the last iteration where the value is not available:
		if(i>1) {
			traits.grad_offset -= lsizes[i-1]*(lsizes[i-2]+1);
		}
	}
}

template <typename T>
void _gd_errfunc(unsigned int nl, unsigned int* lsizes, unsigned int nsamples, 
	T* nn_params, T* X, T* yy, T lambda, T& J, T* gradients, T* deltas, T* inputs)
{
	// Allocate the device memory:
	size_t size;
	hipError_t err;

	// hipDeviceSynchronize();

	// size = nl * sizeof(unsigned int);
	// T* d_lsizes = NULL;
	// checkCudaErrors(hipMalloc(&d_lsizes, size));
	// checkCudaErrors(hipMemcpy(d_lsizes, lsizes, size, hipMemcpyHostToDevice));

	// Compute the total number of parameters in this network:
	unsigned int np = 0;
	unsigned int nt = nl-1; // number of matrices evolved.

	for(unsigned int i=0;i<nt;++i) {
		np += lsizes[i+1]*(lsizes[i]+1);
	}

	size = np * sizeof(T);
	T* d_params = NULL;
	checkCudaErrors(hipMalloc(&d_params, size));
	checkCudaErrors(hipMemcpy(d_params, nn_params, size, hipMemcpyHostToDevice));

	// prepare regularization weigths:
	T* h_regw = new T[size];
	memset(h_regw,0,size);

	// prepare the regularization correction:
	T* rptr = h_regw;

	for(unsigned int i=0; i<nt;++i) {
		unsigned int nrows = lsizes[i+1];
		unsigned int ncolT = lsizes[i]; // we remove 1 here because we consider the intercept row as "virtual" in our calculation.

		rptr += nrows;
		unsigned int count = nrows*ncolT;

		for(unsigned int j=0;j<count;++j) {
			(*rptr++) = 1.0;
		}
	}


	// Prepare the reg weights for this network:
	T* d_regw = NULL;
	checkCudaErrors(hipMalloc(&d_regw, size));

	checkCudaErrors(hipMemcpy(d_regw, h_regw, size, hipMemcpyHostToDevice));

	// Also allocation the gradient array, with the same number of elements:
	T* d_grads = NULL;
	checkCudaErrors(hipMalloc(&d_grads, size));
	checkCudaErrors(hipMemset(d_grads,0,size));

	// Compute the total number of delta coefficients:
	unsigned int nd = 0;
	for(unsigned int i=1;i<nl;++i) {
		nd += lsizes[i]*nsamples;
	}

	size = nd*sizeof(T);
	T* d_deltas = NULL;
	checkCudaErrors(hipMalloc(&d_deltas, size));
	checkCudaErrors(hipMemset(d_deltas,0,size));

	// Prepare the X matrix:
	size = sizeof(T) * nsamples * lsizes[0];
	T* d_X = NULL;
	err = hipMalloc(&d_X, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc X: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_X, X, size, hipMemcpyHostToDevice);

	// Prepare the input data:
	// the size of each input matrix is lsize[i+1]*nsamples;
	// and we need input 0 to nt-1, inclusive.
	// So that's nl input matrices.
	unsigned int count = 0;
	for(unsigned int i=0;i<nt;++i) {
		count += lsizes[i+1];
	}

	size = nsamples * count * sizeof(T);
	size_t input_size = size;
	T* d_inputs = NULL;
	err = hipMalloc(&d_inputs, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc inputs: "<<hipGetErrorString(err));
	}
	hipMemset(d_inputs,0,size); // This is needed for debugging only.

	// Copy the label matrix:	
	size = nsamples * lsizes[nt] * sizeof(T);
	T* d_yy = NULL;
	err = hipMalloc(&d_yy, size);
	if(err!=hipSuccess) {
		logDEBUG("CUDA malloc yy: "<<hipGetErrorString(err));
	}
	hipMemcpy(d_yy, yy, size, hipMemcpyHostToDevice);

	// Call the actual method to perform the computations:
	// costFunc_device(nl, np, lsizes, nsamples, d_params, d_X, d_yy, lambda, J, d_grads, d_deltas, d_inputs, d_regw);
	gd_errfunc_device<T>(nl, np, lsizes, nsamples, d_params, d_X, d_yy, lambda, &J, d_grads, d_deltas, d_inputs, d_regw);

	// Here we should also read back the gradient values:
	checkCudaErrors(hipMemcpy(gradients, d_grads, sizeof(T)*np, hipMemcpyDeviceToHost));
	// memset(gradients,0,sizeof(T)*np);
	
	// Read inputs from device memory
	if(inputs) {
		checkCudaErrors(hipMemcpy(inputs, d_inputs, input_size, hipMemcpyDeviceToHost));
	}

	if(deltas) {
		checkCudaErrors(hipMemcpy(deltas, d_deltas, sizeof(T)*nd, hipMemcpyDeviceToHost)); // only retrieve the deltas if requested.
	}

	// hipDeviceSynchronize();

	// Free device memory
	// checkCudaErrors(hipFree(d_lsizes));
	checkCudaErrors(hipFree(d_params));
	checkCudaErrors(hipFree(d_regw));
	checkCudaErrors(hipFree(d_inputs));	
	checkCudaErrors(hipFree(d_yy));	
	checkCudaErrors(hipFree(d_X));	
	checkCudaErrors(hipFree(d_deltas));	
	checkCudaErrors(hipFree(d_grads));	
	delete [] h_regw;
}

extern "C" {

// void gd_errfunc(unsigned int nl, unsigned int* lsizes, unsigned int nsamples, 
// 	double* nn_params, double* X, double* yy, double lambda, double& J, double* gradients, double* deltas, double* inputs)
void gd_errfunc(BPTraits<double>& traits)
{
		_gd_errfunc(traits.nl, traits.lsizes, traits.nsamples, traits.params, traits.X, 
			traits.yy, traits.lambda, traits.cost, traits.grads, traits.deltas, traits.inputs);
}

void gd_errfunc_f(unsigned int nl, unsigned int* lsizes, unsigned int nsamples, 
	float* nn_params, float* X, float* yy, float lambda, float& J, float* gradients, float* deltas, float* inputs)
{
		_gd_errfunc(nl, lsizes, nsamples, nn_params, X, yy, lambda, J, gradients, deltas, inputs);
}

}
