#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

template<typename T, unsigned int blockSize>
void gd_errfunc_device(BPDeviceTraits<T> &d_traits)
{
  unsigned int nl = d_traits.nl;
  unsigned int np = d_traits.np();
  unsigned int nt = nl - 1; // number of matrices evolved.
  unsigned int nsamples = d_traits.nsamples;
  unsigned int *lsizes = d_traits.lsizes;
  hipStream_t stream = d_traits.stream;

  BPComputeTraits<T> traits;
  traits = d_traits;

  traits.input_offset = nn_activation_device(d_traits);

  T *d_hx = d_traits.inputs + traits.input_offset;

  // Here we can compute the cost now:
  // but only if requested:
  if (d_traits.compute_cost)
  {
    // The hx matrix is mapped to the last z matrix. eg at i=nt-1
    // So its dimensions are lsizes[nt-1+1] * nsamples = lsizes[nl-1] * nsamples
    // same dimensions for the yy matrix, and we want to perform reduction other those 2 matrices
    T J = 0.0;
    unsigned int count = nsamples * lsizes[nt];
    reduce_cost_device(d_hx, d_traits.yy, count, J, stream);

    J /= (T)nsamples;

    T Jreg = 0.0;
    reduce_cost_reg_device(d_traits.params, d_traits.regw, np, Jreg, stream);

    J += (T)((Jreg * d_traits.lambda) / (2.0 * nsamples));
    d_traits.cost = J;
  }

  if (!d_traits.compute_grads)
  {
    // we don't need to compute the gradients.
    return;
  }

  // Prepare the computation of the delta matrices in reverse order:

  // remove the last theta matrix size from the theta offset so that we can use
  // that offset to retrieve the proper theta matrix:
  // theta_offset -= lsizes[nt]*(lsizes[nt-1]+1);
  traits.theta_offset = np - lsizes[nt] * (lsizes[nt - 1] + 1);

  // initially the input_offset is pointing on the hx matrix which is z(nt-1) with our convention (eg. z(0) is not in the array.)
  // But the first one we will need is actually the one before that: z(nt-2)
  // So we need to update the offset, and remove the size of the matrix z(nt-2) ! (pointer is at the beginning of z(nt-1))
  // Note: This is now done inside the loop:
  // input_offset -= lsizes[nt-1]*nsamples;

  // Prepare the offset for the gradient array:
  // keep in mind we start with the latest theta matrix:
  traits.grad_offset = np - lsizes[nt] * (lsizes[nt - 1] + 1);

  for (unsigned int i = nt; i > 0; --i)
  {
    traits.nrows = lsizes[i];
    traits.ncols = nsamples;
    traits.niter = lsizes[i + 1];
    unsigned int count = traits.nrows * traits.ncols;

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((blockSize + traits.ncols - 1) / blockSize, (blockSize + traits.nrows - 1) / blockSize);

    if (i == nt)
    {
      // we should just copy the difference of hx and yy into the z matrix.
      InitLastDelta <<< dimGrid, dimBlock, 0, stream>>>(traits);
    }
    else
    {
      // We compute the delta from the previous delta:
      // We start this computation for delta(nt-1). this matrix is build from theta(nt-1) and delta(nt).
      // also in the process we use the input matrix z(nt-2)
      ComputeDelta <<< dimGrid, dimBlock, 0, stream>>>(traits);

      // once the computation is done for that layer we move to the previous layer:
      traits.theta_offset -= lsizes[i] * (lsizes[i - 1] + 1);
    }

    traits.delta_offset = traits.next_delta_offset;
    traits.next_delta_offset += count;

    // At this point we have the previous theta matrix (eg. theta(i-1) pointed by theta_offset. (both when i=nt and i<nt).
    // and thats the matrix we need to compute the gradient values.
    // the gradient mat has the same size as the current theta matrix.
    // similarly, the input_offset is pointing on z(i-2) which is the one we need to perform the computation too.
    // and delta_offset points to the delta matrix we just wrote (eg. delta(i)).
    traits.nrows = lsizes[i];
    traits.ncols = lsizes[i - 1] + 1;
    traits.niter = nsamples;
    count = traits.nrows * traits.ncols;

    // Also setup the wbias offset to ensure that we start with the correct offset:
    // basically, we have nsamples bias values for each layer, so we just need to multiply
    // nsamples by the desired number of layers offset.
    // When computing the gradients at index i (1<=i<=nt) we need to use the bias from layer (i-1),
    // thus the offset is:
    traits.wbias_offset = (i - 1) * nsamples;

    // Compute the gradient:
    dimBlock = dim3(blockSize, blockSize);
    dimGrid = dim3((blockSize + traits.ncols - 1) / blockSize, (blockSize + traits.nrows - 1) / blockSize);

    traits.input_offset -= lsizes[i - 1] * nsamples; // we remove the size of the next delta matrix to be computed. which is also the size of the next z matrix we will use.
    // logDEBUG("GPU: Gradient at i="<<i<<" of size "<< nrows <<" x " << ncols<<", offset="<<grad_offset<<", input_offset="<<input_offset<<", nsamples="<<nsamples);

    ComputeGradient <<< dimGrid, dimBlock, 0, stream>>>(traits);

    // update the gradient offset by removing the size of the next gradient matrix to be computed:
    // except for the last iteration where the value is not available:
    if (i > 1)
    {
      traits.grad_offset -= lsizes[i - 1] * (lsizes[i - 2] + 1);
    }
  }
}

template <typename T>
void _gd_errfunc(BPTraits<T> &traits)
{
  // BPDeviceTraits<T> d_traits(traits);
  BPDeviceTraits<T> d_traits(false);
  d_traits = traits;

  // Compute the total number of delta coefficients:
  unsigned int nd = traits.nd();
  unsigned int np = traits.np();

  // Call the actual method to perform the computations:
  gd_errfunc_device<T>(d_traits);

  if (traits.compute_cost)
  {
    traits.cost = d_traits.cost;
  }

  // Here we should also read back the gradient values:
  if (traits.compute_grads)
  {
    copyFromDevice(traits.grads, d_traits.grads, np);
  }

  // Read inputs from device memory
  if (traits.inputs)
  {
    copyFromDevice(traits.inputs, d_traits.inputs, nd);
  }

  if (traits.deltas)
  {
    copyFromDevice(traits.deltas, d_traits.deltas, nd); // only retrieve the deltas if requested.
  }
}

template <typename T>
void _gd_errfunc_cpu(BPTraits<T> &traits)
// unsigned int nl, unsigned int* lsizes, unsigned int nsamples,
// double* params, double* X, double* yy, double lambda,
// double* activation, unsigned int ninputs, double* inputs, double& J, double* gradients, double* deltas)
{
  unsigned int nl = traits.nl;
  unsigned int *lsizes = traits.lsizes;
  unsigned int nsamples = traits.nsamples_train;
  unsigned int nt = nl - 1;

  bool owned_inputs = false;
  bool owned_deltas = false;
  bool owned_grads = false;

  if (!traits.inputs)
  {
    owned_inputs = true;
    traits.inputs = new T[traits.nd()];
  }

  if (!traits.deltas)
  {
    owned_deltas = true;
    traits.deltas = new T[traits.nd()];
  }

  if (!traits.grads)
  {
    owned_grads = true;
    traits.grads = new T[traits.np()];
  }

  T *X = traits.X;
  T *yy = traits.yy;
  T *inputs = traits.inputs;
  T *deltas = traits.deltas;
  T *params = traits.params;
  T *gradients = traits.grads;

  T *ptr;

  // First step is to compute the predictions, inside the input array.
  nn_predict_cpu(traits);

  // Compute the value of J on the cpu:

  // Place the input offset at the proper location:
  unsigned int input_offset = 0;
  for (unsigned int i = 1; i < nt; ++i)
  {
    // Add the size of the layer i multiplied by the number of samples:
    input_offset += lsizes[i] * nsamples;
  }

  T *hx = inputs + input_offset;

  if (traits.compute_cost)
  {
    T J = 0.0;

    unsigned int count = nsamples * lsizes[nt];
    for (unsigned int j = 0; j < count; ++j)
    {
      J -= yy[j] * log(hx[j]) + (1.0 - yy[j]) * log(1.0 - hx[j]);
    }

    J /= (double)nsamples;

    // Add the regularisation:
    ptr = params;

    double Jreg = 0.0;
    for (unsigned int j = 0; j < nt; ++j)
    {
      ptr += lsizes[j + 1];
      count = lsizes[j + 1] * (lsizes[j]);
      for (unsigned int k = 0; k < count; ++k)
      {
        double val = (*ptr++);
        Jreg += val * val;
      }
    }

    J += Jreg * traits.lambda / (2.0 * nsamples);

    traits.cost = J;
  }

  if (!traits.compute_grads)
  {
    // we don't need to compute the gradients.
    return;
  }

  // we will now compute the delta vectors:
  // Offset to use when reading the delta matrix in the current iteration
  // except when next_delta_offset is 0, in that case we read the hx and yy matrices.
  unsigned int delta_offset = 0;

  // Offset to use when writing the delta matrix in the current iteration
  unsigned int next_delta_offset = 0;

  // remove the last theta matrix size from the theta offset so that we can use
  // that offset to retrieve the proper theta matrix:
  unsigned int theta_offset = traits.np() - lsizes[nt] * (lsizes[nt - 1] + 1);

  // initially the input_offset is pointing on the hx matrix which is z(nt-1) with our convention (eg. z(0) is not in the array.)
  // But the first one we will need is actually the one before that: z(nt-2)
  // So we need to update the offset, and remove the size of the matrix z(nt-2) ! (pointer is at the beginning of z(nt-1))
  // Note that this is now done inside the loop.
  // input_offset -= lsizes[nt-1]*nsamples;

  // Prepare the offset for the gradient array:
  // keep in mind we start with the latest theta matrix:
  unsigned int grad_offset = traits.np() - lsizes[nt] * (lsizes[nt - 1] + 1);

  ptr = traits.deltas;

  for (unsigned int i = nt; i > 0; --i)
  {
    unsigned int nrows = lsizes[i];
    unsigned int ncols = nsamples;
    unsigned int niter = lsizes[i + 1];
    unsigned int count = nrows * ncols;

    if (i == nt)
    {
      // We just write the difference of hx and yy in the deltas array:
      for (unsigned int j = 0; j < count; ++j)
      {
        (*ptr++) = hx[j] - yy[j];
      }
    }
    else
    {
      for (unsigned int c = 0; c < ncols; ++c)
      {
        for (unsigned int r = 0; r < nrows; ++r)
        {
          // we want to compute the value delta(r,c);
          double val = 0.0;
          for (unsigned int n = 0; n < niter; ++n)
          {
            // val += theta_T(r+1,n)*delta_prev(n,c);
            // val += theta(n,r+1)*delta_prev(n,c);
            val += params[theta_offset + niter * (r + 1) + n] * deltas[delta_offset + niter * c + n];
          }

          // Then we multiply by the sigmoid gradient at z(r,c):
          double sig = inputs[input_offset + nrows * c + r];
          // deltas[next_delta_offset + nrows*c + r] = next_delta_offset + nrows*c + r;
          deltas[next_delta_offset + nrows * c + r] = val * sig * (1.0 - sig);
        }
      }

      // once the computation is done for that layer we move to the previous layer:
      theta_offset -= lsizes[i] * (lsizes[i - 1] + 1);
    }

    delta_offset = next_delta_offset;
    next_delta_offset += count;

    // At this point we have the previous theta matrix (eg. theta(i-1) pointed by theta_offset. (both when i=nt and i<nt).
    // and thats the matrix we need to compute the gradient values.
    // the gradient mat has the same size as the current theta matrix.
    // similarly, the input_offset is pointing on z(i-2) which is the one we need to perform the computation too.
    // and delta_offset points to the delta matrix we just wrote (eg. delta(i)).
    nrows = lsizes[i];
    ncols = lsizes[i - 1] + 1;
    niter = nsamples;
    count = nrows * ncols;

    input_offset -= lsizes[i - 1] * nsamples; // we remove the size of the next delta matrix to be computed. which is also the size of the next z matrix we will use.
    // logDEBUG("CPU: Gradient at i="<<i<<" of size "<< nrows <<" x " << ncols<<", offset="<<grad_offset<<", input_offset="<<input_offset);

    // Compute the gradient:
    for (unsigned int c = 0; c < ncols; ++c)
    {
      for (unsigned int r = 0; r < nrows; ++r)
      {
        // we want to compute the value of the gradient matrix mat(r,c)
        // with mat_i = delta_i * act_i-1.
        double val = 0.0;
        for (unsigned int n = 0; n < nsamples; ++n)
        {
          // val += delta(r,n)*act(n,c);
          // if c==0 then act[i-1](n,c)==1 otherwise act[i-1](n,c)=z[i-2]_T(n,c-1)=z[i-2](c-1,n)
          // val += deltas[delta_offset + nrows*n + r]; //*(c==0 ? 1.0 : inputs[input_offset + (ncols-1)*n + c-1 ]);
          if (i == 1)
          {
            // Here we have to use the X matrix instead of the z_T.
            // we still want to write the value act(n,c)=x(n,c-1) if c>0
            val += deltas[delta_offset + nrows * n + r] * (c == 0 ? 1.0 : X[niter * (c - 1) + n]);
          }
          else
          {
            if (c == 0)
            {
              val += deltas[delta_offset + nrows * n + r];
            }
            else
            {
              int index = input_offset + (ncols - 1) * n + c - 1;
              val += deltas[delta_offset + nrows * n + r] * (c == 0 ? 1.0 : inputs[index]);
            }
          }
          // val += 1.0; //(c==0 ? 1.0 : inputs[input_offset + (ncols-1)*n + c-1 ]);
        }

        // Here we also need to add the regularization from the theta matrix:
        double reg = (c == 0 ? 0.0 : params[theta_offset + nrows * c + r]);
        val += traits.lambda * reg;

        gradients[grad_offset + nrows * c + r] = val / niter; //grad_offset + nrows*c + r; //val/niter;
      }
    }

    // update the gradient offset by removing the size of the next gradient matrix to be computed:
    // except for the last iteration where the value is not available:
    if (i > 1)
    {
      grad_offset -= lsizes[i - 1] * (lsizes[i - 2] + 1);
    }
  }

  // release local resources:
  if (owned_inputs)
  {
    delete [] traits.inputs;
    traits.inputs = nullptr;
  }

  if (owned_deltas)
  {
    delete [] traits.deltas;
    traits.deltas = nullptr;
  }

  if (owned_grads)
  {
    delete [] traits.grads;
    traits.grads = nullptr;
  }
}

extern "C" {

  void gd_errfunc(BPTraits<double> &traits)
  {
    _gd_errfunc(traits);
  }

  void gd_errfunc_f(BPTraits<float> &traits)
  {
    _gd_errfunc(traits);
  }

  void gd_errfunc_cpu(BPTraits<double> &traits)
  {
    _gd_errfunc_cpu(traits);
  }

}
