#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

template<typename T, unsigned int blockSize>
int nn_activation_device(BPDeviceTraits<T> &d_traits)
{
  unsigned int nt = d_traits.nl - 1; // number of matrices evolved.
  unsigned int *lsizes = d_traits.lsizes;
  unsigned int nsamples = d_traits.nsamples;
  T *wmults = d_traits.wmults;
  T *dropouts = d_traits.dropouts;

  hipStream_t stream = d_traits.stream;

  RandDeviceTraits<T> r_traits;
  r_traits.randStates = d_traits.randStates;
  r_traits.debug = d_traits.debug;

  // Assign the wX buffer before creating any ComputeTraits:
  if (!dropouts)
  {
    d_traits.wX = d_traits.X;
  }
  else
  {
    // Assign a random drop for the wX buffer:
    THROW_IF(!d_traits.rX,"Invalid rX buffer.");

    r_traits.target = d_traits.rX;
    r_traits.size = lsizes[0] * nsamples;
    r_traits.values = d_traits.X;
    r_traits.threshold = dropouts[0];

    rand_weights_device(r_traits);
    
    r_traits.values = nullptr;

    d_traits.wX = d_traits.rX; 
  }

  // Ensure we have the proper generic settings for the rand weights computation:
  r_traits.size = nsamples;
  r_traits.value = d_traits.bias;

  BPComputeTraits<T> traits;
  traits = d_traits;


  for (unsigned int i = 0; i < nt; ++i)
  {
    // We compute the activation and input values for the given layer:

    // The kernel compute the values of zi and a(i+1)
    // (note that the value or a(0) is already loaded in the Activation vector).
    // even if we compute the a(i+1) matrix we actually discard completely the first column
    // in this matrix (colu of intercept terms). As a result we just need to mapped the GPU grid to
    // the dimension of of the sub z(i) matrix (which is transposed.)
    // THe dimensions for z(i) are: lsize(i+1) * nsamples
    // When this is transposed we get: nsamples * lsize(i+1);
    unsigned int nrows = lsizes[i + 1];
    unsigned int ncolT = lsizes[i]; // we remove 1 here because we consider the intercept row as "virtual" in our calculation.
    unsigned int ncols = nsamples;

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((blockSize + ncols - 1) / blockSize, (blockSize + nrows - 1) / blockSize);
    // logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");

    traits.nrows = nrows;
    traits.ncols = ncols;
    traits.niter = ncolT;

    if (wmults)
      traits.wmult = wmults[i];

    // Also we will need access to the theta_i matrix so we need to keep track of its global offset in the
    // network parameters array.
    if (dropouts)
    {
      traits.layer_dropout = i == (nt - 1) ? (T)1.0 : dropouts[i + 1]; // we don't want to drop anything from the output layer.

      r_traits.target = traits.wbias + traits.wbias_offset;
      r_traits.threshold = dropouts[i];

      rand_weights_device(r_traits);

      // Update the bias weights to be used for this layer computation:
      if (d_traits.debug)
      {
        ComputeActivation<T, true, true> <<< dimGrid, dimBlock, 0, stream>>>(traits);
      }
      else
      {
        // use really random weights:
        ComputeActivation<T, true> <<< dimGrid, dimBlock, 0, stream>>>(traits);
      }
    }
    else
    {
      ComputeActivation <<< dimGrid, dimBlock, 0, stream>>>(traits);
    }

    // update the offsets:
    traits.wbias_offset += ncols;
    traits.theta_offset += lsizes[i + 1] * (lsizes[i] + 1);
    traits.input_offset = traits.next_input_offset;
    traits.next_input_offset += nrows * ncols;
  }

  return traits.input_offset;
}


template <typename T>
void _nn_predict(BPTraits<T> &traits)
{
  BPDeviceTraits<T> d_traits;
  d_traits = traits;

  int input_offset = nn_activation_device(d_traits);

  if (traits.hx)
  {
    copyFromDevice(traits.hx, d_traits.inputs + input_offset, traits.ny());
    // copyFromDevice(traits.hx, d_traits.predictions(), traits.ny());
  }
}

template <typename T>
void _nn_predict_cpu(BPTraits<T> &traits)
{
  unsigned int nl = traits.nl;
  unsigned int *lsizes = traits.lsizes;
  unsigned int nsamples = traits.nsamples_train;
  T *params = traits.params;
  T *X = traits.X;
  T *hx = traits.hx;
  T bias = traits.bias;
  T *wmults = traits.wmults;

  // method used to compute the activation on the CPU.
  unsigned int nt = nl - 1;

  // Compute the number of parameters:
  // and compute the number of activation (eg. inputs) values:
  unsigned int np = 0;
  unsigned int ni = 0;
  for (unsigned int i = 0; i < nt; ++i)
  {
    np += lsizes[i + 1] * (lsizes[i] + 1);
    ni += lsizes[i + 1] * nsamples;
  }

  // Prepare the input array:
  T *inputs = traits.inputs;
  if (!inputs)
  {
    inputs = new T[ni];
  }

  // offset used to locate the theta_i matrix in the d_params array.
  unsigned int theta_offset = 0;

  // Offset used for the z(i) matrix on iteration i
  int input_offset = 0;

  int next_input_offset = 0; //nsamples*lsizes[1];
  T mult = 1.0; // default weight multiplier value.

  for (unsigned int i = 0; i < nt; ++i)
  {
    // We compute the activation and input values for the given layer:

    unsigned int nrows = lsizes[i + 1];
    unsigned int ncolT = lsizes[i]; // we remove 1 here because we consider the intercept row as "virtual" in our calculation.
    unsigned int ncols = nsamples;

    // Check if a weight multiplier is provided for this layer:
    if (wmults)
      mult = wmults[i];

    T xw;

    for (unsigned int c = 0; c < ncols; ++c)
    {
      for (unsigned int r = 0; r < nrows; ++r)
      {
        // compute the activation on unit r, for sample c;
        T bval = bias;
        if (traits.dropouts && (abs(sin(c)) > traits.dropouts[i]))
        {
          // Flags the bias value with 0.0 if we should ignore that unit:
          bval = 0.0;
        }

        T val = bval * params[theta_offset + r];

        for (unsigned int j = 0; j < ncolT; ++j)
        {
          // Add the element theta(r,j+1)*a(j,c)
          // if we are on i==0, then we are using X as activation, and in that case
          // we need to transpose the value:
          if (i == 0)
          {
            xw = 1.0;
            if(traits.dropouts && (abs(sin(nsamples * j + c)) > traits.dropouts[0]))
            {
              xw = 0.0;
            }

            val += params[theta_offset + nrows * (j + 1) + r] * X[nsamples * j + c] * xw;
          }
          else
          {
            val += params[theta_offset + nrows * (j + 1) + r] * inputs[input_offset + ncolT * c + j];
          }
        }

        // Now assign the computed value to the input array:
        // The compute value is a(r,c)
        T zval = (T)(1.0 / (1.0 + exp(-val * mult)));

        if (traits.dropouts)
        {
          T drop = 1.0;
          if (i < (nt - 1))
          {
            drop = traits.dropouts[i + 1];
          }

          if (abs(sin(nrows * c + r)) > drop)
          {
            zval = 0.0;
          }
        }

        inputs[next_input_offset + nrows * c + r] = zval;
      }
    }

    // update the offsets:
    theta_offset += lsizes[i + 1] * (lsizes[i] + 1);
    input_offset = next_input_offset;
    next_input_offset += nrows * ncols;
  }

  // Now we need to copy the last input data in the hx matrix:
  if (hx)
  {
    memcpy(hx, inputs + input_offset, lsizes[nt]*nsamples * sizeof(T));
  }

  // Delete if we have ownership:
  if (!traits.inputs)
  {
    delete [] inputs;
  }
}

extern "C" {

  void nn_predict(BPTraits<double> &traits)
  {
    _nn_predict(traits);
  }

  void nn_predict_f(BPTraits<float> &traits)
  {
    _nn_predict(traits);
  }

  void nn_predict_cpu(BPTraits<double> &traits)
  {
    _nn_predict_cpu(traits); //nl, lsizes, nsamples, params, X, hx, bias, wmults);
  }

  void nn_predict_cpu_f(BPTraits<float> &traits)
  {
    _nn_predict_cpu(traits); //nl, lsizes, nsamples, params, X, hx, bias, wmults);
  }

}
