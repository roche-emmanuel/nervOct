#include <nervCUDA.h>
#include <nerv_kernels.h>
#include "hipblas.h"

// Method used to compute op(A)*x using CUBLAS:
template<typename T>
void mat_vec_mult_device(hipblasHandle_t handle, hipblasOperation_t trans, unsigned int nrows, unsigned int ncols, T *A, T* x, T* y)
{
	T alpha = (T)1;
	hipblasSgemv(handle, trans, nrows, ncols, &alpha, A, nrows, x, 1, NULL, y, 1);
}

template<>
void mat_vec_mult_device<double>(hipblasHandle_t handle, hipblasOperation_t trans, unsigned int nrows, unsigned int ncols, double *A, double* x, double* y)
{
	double alpha = (double)1;
	hipblasDgemv(handle, trans, nrows, ncols, &alpha, A, nrows, x, 1, NULL, y, 1);
}

template <typename T>
void _mat_vec_mult(unsigned int nrows, unsigned int ncols, T *A, T* x, T* y, bool tpA)
{
  size_t size;

  size = nrows*ncols * sizeof(T);
  T *d_A = NULL;
  checkCudaErrors(hipMalloc(&d_A, size));
  checkCudaErrors(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));

  size = (tpA ? nrows : ncols) * sizeof(T);
  T *d_x = NULL;
  checkCudaErrors(hipMalloc(&d_x, size));
  checkCudaErrors(hipMemcpy(d_x, x, size, hipMemcpyHostToDevice));

  size = (tpA ? ncols : nrows) * sizeof(T);
  T *d_y = NULL;
  checkCudaErrors(hipMalloc(&d_y, size));
  // checkCudaErrors(hipMemcpy(d_y, vec2, size, hipMemcpyHostToDevice));

  hipblasHandle_t handle;
	checkCublasErrors(hipblasCreate(&handle));

	// hipStream_t stream;
	// checkCublasErrors(hipblasSetStream(handle, stream));

  mat_vec_mult_device(handle, tpA ? HIPBLAS_OP_T : HIPBLAS_OP_N, nrows, ncols, d_A, d_x, d_y);

	checkCublasErrors(hipblasDestroy(handle));

  copyFromDevice(y, d_y, (tpA ? ncols : nrows));

  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_y));
}

extern "C" {

  void mat_vec_mult(unsigned int nrows, unsigned int ncols, double *A, double* x, double* y, bool tpA)
  {
    _mat_vec_mult(nrows,ncols,A,x,y,tpA);
  }

  void mat_vec_mult_f(unsigned int nrows, unsigned int ncols, float *A, float* x, float* y, bool tpA)
  {
    _mat_vec_mult(nrows,ncols,A,x,y,tpA);
  }

}
