#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

/*
    This version adds multiple elements per thread sequentially.  This reduces the overall
    cost of the algorithm while keeping the work complexity O(n) and the step complexity O(log n).
    (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void reduce6(T *g_idata, T *g_odata, unsigned int n)
{
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = mySum;
}


////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template<typename T>
void reduce_sum_launcher(int size, int threads, int blocks, int whichKernel, T *d_idata, T *d_odata)
{
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

  // choose which of the optimized versions of reduction to launch
  if (isPow2(size))
  {
      switch (threads)
      {
          case 512:
              reduce6<T, 512, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 256:
              reduce6<T, 256, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 128:
              reduce6<T, 128, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 64:
              reduce6<T,  64, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 32:
              reduce6<T,  32, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 16:
              reduce6<T,  16, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  8:
              reduce6<T,   8, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  4:
              reduce6<T,   4, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  2:
              reduce6<T,   2, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  1:
              reduce6<T,   1, true><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;
      }
  }
  else
  {
      switch (threads)
      {
          case 512:
              reduce6<T, 512, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 256:
              reduce6<T, 256, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 128:
              reduce6<T, 128, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 64:
              reduce6<T,  64, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 32:
              reduce6<T,  32, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case 16:
              reduce6<T,  16, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  8:
              reduce6<T,   8, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  4:
              reduce6<T,   4, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  2:
              reduce6<T,   2, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;

          case  1:
              reduce6<T,   1, false><<< dimGrid, dimBlock, smemSize >>>(d_idata, d_odata, size);
              break;
      }
  }
}

template <typename T>
void _reduce_sum(T* inputs, unsigned int n, T& output)
{
  int maxThreads = 256;
  int maxBlocks = 64;
  int whichKernel = 6;
  // bool cpuFinalReduction = false;
  int cpuFinalThreshold = 1;

  int numBlocks = 0;
  int numThreads = 0;
  getNumBlocksAndThreads(whichKernel, n, maxBlocks, maxThreads, numBlocks, numThreads);


  // Allocate the input array:
  size_t size = n * sizeof(T);
  T* d_idata = NULL;
  checkCudaErrors(hipMalloc(&d_idata, size));
  checkCudaErrors(hipMemcpy(d_idata, inputs, size, hipMemcpyHostToDevice));

  // Allocate output array:
  size = numBlocks*sizeof(T);
  T* d_odata = NULL;
  checkCudaErrors(hipMalloc(&d_odata, size));
  checkCudaErrors(hipMemcpy(d_odata, inputs, size, hipMemcpyHostToDevice));

  // Allocate mem for the result on host side
  T *h_odata = (T *) malloc(numBlocks*sizeof(T));

  // warm-up
  // reduce<T>(size, numThreads, numBlocks, whichKernel, d_idata, d_odata);

  T gpu_result = 0.0;
  bool needReadBack = true;

  // execute the kernel
  reduce_sum_launcher(n, numThreads, numBlocks, whichKernel, d_idata, d_odata);

  // sum partial block sums on GPU
  int s=numBlocks;
  int kernel = whichKernel;

  while (s > cpuFinalThreshold)
  {
      int threads = 0, blocks = 0;
      getNumBlocksAndThreads(kernel, s, maxBlocks, maxThreads, blocks, threads);

      reduce_sum_launcher(s, threads, blocks, kernel, d_odata, d_odata);

      if (kernel < 3)
      {
          s = (s + threads - 1) / threads;
      }
      else
      {
          s = (s + (threads*2-1)) / (threads*2);
      }
  }

  if (s > 1)
  {
      // copy result from device to host
      checkCudaErrors(hipMemcpy(h_odata, d_odata, s * sizeof(T), hipMemcpyDeviceToHost));

      for (int i=0; i < s; i++)
      {
          gpu_result += h_odata[i];
      }

      needReadBack = false;
  }

  if (needReadBack)
  {
      // copy final sum from device to host
      checkCudaErrors(hipMemcpy(&gpu_result, d_odata, sizeof(T), hipMemcpyDeviceToHost));
  }

  // store the result:
  output = gpu_result;

  // Free host memory:
  free(h_odata);

  // Free device memory
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));
}

extern "C" {

void reduce_sum(double* inputs, unsigned int n, double& output)
{
  _reduce_sum(inputs, n, output);
}

void reduce_sum_f(float* inputs, unsigned int n, float& output)
{
  _reduce_sum(inputs, n, output);
}

}
