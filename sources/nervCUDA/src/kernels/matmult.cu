#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

template<typename T, unsigned int blockSize>
__global__ void MatMult(unsigned int nrowA, unsigned int niter, unsigned int ncolB, const T* A, const T* B, T* C) {

	// Note that we assume here that the matrix coefficient are stored in row major order:
	// eg Aelem(i,jl) = A[j*nrowA+i]
  T CValue = 0;

  __shared__ T As[blockSize][blockSize+1]; // Adding +1 to avoid shared memory bank conflict
  __shared__ T Bs[blockSize][blockSize+1];

  int xx, yy;
  for (int k = 0; k < (blockSize + niter - 1)/blockSize; k++) {

  	// Here we try to access the A matrix data in a coaleased way:
  	// keeping in mind that A is row major. So we need to read A per column
  	// while the threads in the wrap are (probably) organized by row.
  	// So we invert the roles palyed by threadIdx.x and threadIdx.y.
  	xx = k*blockSize + threadIdx.y;
  	yy = blockIdx.y*blockSize + threadIdx.x;
		if (xx < niter && yy < nrowA) 
		 	As[threadIdx.x][threadIdx.y] = A[xx*nrowA + yy];
		else
			As[threadIdx.x][threadIdx.y] = 0.0;


		// Same for the B matrix, we need to invert the x and y coords:
		xx = blockIdx.x*blockSize + threadIdx.y;
		yy = k*blockSize + threadIdx.x;

		if (yy < niter && xx < ncolB)
			Bs[threadIdx.x][threadIdx.y] = B[xx*niter + yy];
		else
			Bs[threadIdx.x][threadIdx.y] = 0.0;

		__syncthreads();

		for (int n = 0; n < blockSize; ++n) 
			CValue += As[threadIdx.x][n] * Bs[n][threadIdx.y];

		__syncthreads();
  }

  int row = blockIdx.y*blockSize + threadIdx.x;
  int col = blockIdx.x*blockSize + threadIdx.y;

  if (row < nrowA && col < ncolB)
  	C[col*nrowA + row] = CValue;
}

template<typename T, unsigned int blockSize>
__global__ void MatMultTpA(unsigned int nrowC, unsigned int niter, unsigned int ncolC, const T* A, const T* B, T* C) {

	// Note that we assume here that the matrix coefficient are stored in row major order:
	// eg Aelem(i,jl) = A[j*nrowA+i]
  T CValue = 0;

  int row = blockIdx.y*blockSize + threadIdx.y;
  int col = blockIdx.x*blockSize + threadIdx.x;

  __shared__ T As[blockSize][blockSize+1];
  __shared__ T Bs[blockSize][blockSize+1];

  int xx, yy;
  for (int k = 0; k < (blockSize + niter - 1)/blockSize; k++) {

  	xx = k*blockSize + threadIdx.x;
  	yy = row;
  	
		if (yy < nrowC && xx < niter) 
		 	As[threadIdx.y][threadIdx.x] = A[yy*niter + xx];
		else
			As[threadIdx.y][threadIdx.x] = 0.0;

		xx = blockIdx.x*blockSize + threadIdx.y;
		yy = k*blockSize + threadIdx.x;

		if (yy < niter && xx < ncolC)
			Bs[threadIdx.x][threadIdx.y] = B[xx*niter + yy];
		else
			Bs[threadIdx.x][threadIdx.y] = 0.0;

		__syncthreads();

		for (int n = 0; n < blockSize; ++n) 
			CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

		__syncthreads();
  }

  if (row < nrowC && col < ncolC)
  	C[ (blockIdx.x*blockSize + threadIdx.x)*nrowC + blockIdx.y*blockSize+threadIdx.y] = CValue;
}

template<typename T, unsigned int blockSize>
__global__ void MatMultTpB(unsigned int nrowC, unsigned int niter, unsigned int ncolC, const T* A, const T* B, T* C) 
{
	// unsigned int nrowA, unsigned int ncolA,
 //    unsigned int nrowB, unsigned int ncolB,) {

	// Note that we assume here that the matrix coefficient are stored in row major order:
	// eg Aelem(i,jl) = A[j*nrowA+i]
  T CValue = 0;

  int row = blockIdx.y*blockSize + threadIdx.y;
  int col = blockIdx.x*blockSize + threadIdx.x;

  __shared__ T As[blockSize][blockSize+1];
  __shared__ T Bs[blockSize][blockSize+1];

  int xx, yy;

  for (int k = 0; k < (blockSize + niter - 1)/blockSize; k++) {

  	xx = k*blockSize + threadIdx.y;
  	yy = blockIdx.y*blockSize + threadIdx.x;

		if (xx < niter && yy < nrowC) 
		 	As[threadIdx.x][threadIdx.y] = A[xx*nrowC + yy];
		else
			As[threadIdx.x][threadIdx.y] = 0.0;

		xx = col;
		yy = k*blockSize + threadIdx.y;

		if (xx < ncolC && yy < niter)
			Bs[threadIdx.y][threadIdx.x] = B[yy*ncolC + xx];
		else
			Bs[threadIdx.y][threadIdx.x] = 0.0;

		__syncthreads();

		for (int n = 0; n < blockSize; ++n) 
			CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

		__syncthreads();
  }

  if (row < nrowC && col < ncolC)
  	C[ (blockIdx.x*blockSize + threadIdx.x)*nrowC + blockIdx.y*blockSize+threadIdx.y] = CValue;
}

template<typename T>
void matmult_device(unsigned int nrowA, unsigned int ncolA, unsigned int nrowB, unsigned int ncolB, 
	const T* d_A, const T* d_B, T* d_C, bool tpA, bool tpB)
{
	// Call the kernel directly:
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid((BLOCK_SIZE + (tpB ? nrowB : ncolB)-1)/BLOCK_SIZE, (BLOCK_SIZE + (tpA ? ncolA : nrowA)-1)/BLOCK_SIZE);
	
	// logDEBUG("Using grid size: ("<<dimGrid.x<<" x "<<dimGrid.y<<")");
	if(tpA) {
		MatMultTpA<<<dimGrid, dimBlock>>>(ncolA, nrowA, ncolB, d_A, d_B, d_C);
	}
	else if(tpB) {
		MatMultTpB<<<dimGrid, dimBlock>>>(nrowA, ncolA, nrowB, d_A, d_B, d_C);
	}
	else {
		MatMult<<<dimGrid, dimBlock>>>(nrowA, ncolA, ncolB, d_A, d_B, d_C);
	}
}

template<typename T>
void _matmult(unsigned int nrowA, unsigned int ncolA, const T* A,
    unsigned int nrowB, unsigned int ncolB, const T* B, T* C, bool tpA, bool tpB)
{
	// Allocate the device memory:
	size_t size;

	size = nrowA * ncolA * sizeof(T);
	T* d_A = NULL;
	checkCudaErrors(hipMalloc(&d_A, size));
	checkCudaErrors(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));

	size = nrowB * ncolB * sizeof(T);
	T* d_B = NULL;
	checkCudaErrors(hipMalloc(&d_B, size));
	checkCudaErrors(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

	size = (tpA ? ncolA : nrowA) * (tpB ? nrowB : ncolB) * sizeof(T);
	T* d_C = NULL;
	checkCudaErrors(hipMalloc(&d_C, size));
	// hipMemcpy(d_B, B, size, hipMemcpyHostToDevice); // no need to set this.

	matmult_device(nrowA, ncolA, nrowB, ncolB, d_A, d_B, d_C, tpA, tpB);

	// Read C from device memory
	checkCudaErrors(hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost));
	// logDEBUG("Copy C off of device: "<<hipGetErrorString(err));

	// Free device memory
	checkCudaErrors(hipFree(d_A));
	checkCudaErrors(hipFree(d_B));
	checkCudaErrors(hipFree(d_C));
}

extern "C" {


void matmult(unsigned int nrowA, unsigned int ncolA, const double* A,
    unsigned int nrowB, unsigned int ncolB, const double* B, double* C, bool tpA, bool tpB)
{
	_matmult(nrowA,ncolA,A,nrowB,ncolB,B,C,tpA,tpB);
}

void matmult_f(unsigned int nrowA, unsigned int ncolA, const float* A,
    unsigned int nrowB, unsigned int ncolB, const float* B, float* C, bool tpA, bool tpB)
{
	_matmult(nrowA,ncolA,A,nrowB,ncolB,B,C,tpA,tpB);
}

}
