#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

template<typename T, unsigned int blockSize>
__global__ void InitLastDelta(BPComputeTraits<T> traits)
// unsigned int input_offset, unsigned int nrows, unsigned int ncols, T* deltas, T* inputs, T* yy) 
{
  int row = blockIdx.y*blockSize + threadIdx.x; // we inverse x and y for coalesced global memory access
  int col = blockIdx.x*blockSize + threadIdx.y;

  unsigned int nrows = traits.nrows;
  unsigned int ncols = traits.ncols;

  if (row < nrows && col < ncols) {
	  int index = nrows*col+row;
   	traits.deltas[index] = traits.inputs[traits.input_offset+index] - traits.yy[index];
  }
}

// Explicit instanciation:
template __global__ void InitLastDelta(BPComputeTraits<double> traits);

template __global__ void InitLastDelta(BPComputeTraits<float> traits);
