#include "hip/hip_runtime.h"
#include <nervCUDA.h>
#include <nerv_kernels.h>

template <typename T, unsigned int blockSize>
__global__ void ComputeActivation(unsigned int theta_offset, unsigned int input_offset, unsigned int next_input_offset,
	unsigned int nrows, unsigned int ncols, unsigned int ncolT, T* nn_params, T* inputs, T* X, T bias, T wmult) 
{

	// Note that we assume here that the matrix coefficient are stored in row major order:
	// eg Aelem(i,jl) = A[j*nrowA+i]
  int row = blockIdx.y*blockSize + threadIdx.x;
  int col = blockIdx.x*blockSize + threadIdx.y;

  __shared__ T As[blockSize][blockSize+1];
  __shared__ T Bs[blockSize][blockSize+1];

  int xx, yy;

  // we can already add the element on the first row of theta_i to this element value:
  // but note that this element should be multiplied with the desired bias:
  T zval = nn_params[theta_offset + row]*bias;

  // Here we compute the product theta_i * a_i^T
  for (int k = 0; k < (blockSize + ncolT - 1)/blockSize; k++) {

  	xx = k*blockSize + threadIdx.y;
  	yy = blockIdx.y*blockSize + threadIdx.x;

		if (xx < ncolT && yy < nrows) 
			// Note here that we should NOT use the first row of theta_i in those computation:
			// That row elemtn is already added to the zval value (matching the "virtual" 1 row
			// on top of the z_i matrix when used as activation.)
		 	As[threadIdx.x][threadIdx.y] = nn_params[theta_offset + (xx+1)*nrows + yy];
		else
			As[threadIdx.x][threadIdx.y] = 0.0;


		if(next_input_offset==0) {
			// In that case we need to retrieve the data from the X matrix.
			// actually we need the data from X^T.
			xx = blockIdx.x*blockSize + threadIdx.x;
			yy = k*blockSize + threadIdx.y;

			if (xx < ncols && yy < ncolT)
				Bs[threadIdx.y][threadIdx.x] = X[yy*ncols + xx];
			else
				Bs[threadIdx.y][threadIdx.x] = 0.0;
		}
		else {
			xx = blockIdx.x*blockSize + threadIdx.y;
			yy = k*blockSize + threadIdx.x;

			if (yy < ncolT && xx < ncols)
				Bs[threadIdx.x][threadIdx.y] = inputs[input_offset + xx*ncolT + yy];
			else
				Bs[threadIdx.x][threadIdx.y] = 0.0;
		}

		__syncthreads();

		for (int n = 0; n < blockSize; ++n) 
			zval += As[threadIdx.x][n] * Bs[n][threadIdx.y];

		__syncthreads();
  }

  if (row < nrows && col < ncols) {
  	// compute the sigmoid of the value:
  	zval = 1.0 / (1.0 + exp(-zval*wmult));

  	// we just computed the value z_i(row,col), now we store it:
  	inputs[next_input_offset + nrows*col + row] = zval;
  }

}

// Explicit specialization:
template __global__ void ComputeActivation<double>(unsigned int theta_offset, unsigned int input_offset, unsigned int next_input_offset,
	unsigned int nrows, unsigned int ncols, unsigned int ncolT, double* nn_params, double* inputs, double* X, double bias, double wmult);

template __global__ void ComputeActivation<float>(unsigned int theta_offset, unsigned int input_offset, unsigned int next_input_offset,
	unsigned int nrows, unsigned int ncols, unsigned int ncolT, float* nn_params, float* inputs, float* X, float bias, float wmult);
